#include <stdio.h>
#include <stdlib.h>
#include "util.h"

#define MAX_FILE_SIZE 5368709120


// Define this to turn on error checking
#define CUDA_ERROR_CHECK

#define CudaSafeCall( err ) __cudaSafeCall( err, __FILE__, __LINE__ )
#define CudaCheckError()    __cudaCheckError( __FILE__, __LINE__ )

inline void __cudaSafeCall( hipError_t err, const char *file, const int line )
{
#ifdef CUDA_ERROR_CHECK
    if ( hipSuccess != err )
    {
        fprintf( stderr, "cudaSafeCall() failed at %s:%i : %s\n",
                 file, line, hipGetErrorString( err ) );
        exit( -1 );
    }
#endif

    return;
}

inline void __cudaCheckError( const char *file, const int line )
{
#ifdef CUDA_ERROR_CHECK
    hipError_t err = hipGetLastError();
    if ( hipSuccess != err )
    {
        fprintf( stderr, "cudaCheckError() failed at %s:%i : %s\n",
                 file, line, hipGetErrorString( err ) );
        exit( -1 );
    }

    // More careful checking. However, this will affect performance.
    // Comment away if needed.
    err = hipDeviceSynchronize();
    if( hipSuccess != err )
    {
        fprintf( stderr, "cudaCheckError() with sync failed at %s:%i : %s\n",
                 file, line, hipGetErrorString( err ) );
        exit( -1 );
    }
#endif

    return;
}



int main()
{
	//host data
	int i;
	char* data;
	unsigned char *hash_table, *final_hash;
	float elapsed1=0, elapsed2 = 0;

	//device data
	char* d_data;
	unsigned char* d_hash;
	int* d_filesize;

	//time measurement
	hipEvent_t start, stop;
	CudaSafeCall(hipEventCreate(&start));
	CudaSafeCall(hipEventCreate(&stop));


	FILE* fp = fopen("test_file.txt", "r");
	if(fp==NULL)
	{
		printf("No such file exists\n");
		exit(0);
	}
	size_t filesize;
	data = (char*) malloc(sizeof(char) * MAX_FILE_SIZE);
	filesize = fread(data, sizeof(char), MAX_FILE_SIZE/sizeof(char), fp);



	CudaSafeCall(hipMalloc((void**) &d_data, filesize));
	CudaSafeCall(hipMalloc((void**) &d_hash, NTHREAD*MD5_DIGEST_LENGTH*sizeof(unsigned char)));
	CudaSafeCall(hipMalloc((void**) &d_filesize, sizeof(int)));
	CudaSafeCall(hipMemcpy(d_data, data, filesize, hipMemcpyHostToDevice));
	CudaSafeCall(hipMemcpy(d_filesize, &filesize, sizeof(int), hipMemcpyHostToDevice));

	CudaSafeCall(hipEventRecord(start, 0));
	hash_blocks_intra<<<1,NTHREAD>>>(d_data, d_hash, d_filesize);
	CudaCheckError();
	CudaSafeCall(hipEventRecord(stop, 0));
	CudaSafeCall(hipEventSynchronize (stop) );
	CudaSafeCall(hipEventElapsedTime(&elapsed1, start, stop) );

	hash_table = (unsigned char*) malloc(NTHREAD*MD5_DIGEST_LENGTH*sizeof(unsigned char));
	CudaSafeCall(hipMemcpy(hash_table, d_hash, NTHREAD*MD5_DIGEST_LENGTH*sizeof(unsigned char), hipMemcpyDeviceToHost));


	CudaSafeCall(hipEventRecord(start, 0));
	hash_blocks_inter<<<1,1>>>(d_hash);
	CudaCheckError();
	CudaSafeCall(hipEventRecord(stop, 0));
	CudaSafeCall(hipEventSynchronize (stop) );
	CudaSafeCall(hipEventElapsedTime(&elapsed2, start, stop) );

	final_hash = (unsigned char*) malloc(MD5_DIGEST_LENGTH*sizeof(unsigned char));
	CudaSafeCall(hipMemcpy(hash_table, d_hash, NTHREAD*MD5_DIGEST_LENGTH*sizeof(unsigned char), hipMemcpyDeviceToHost));

	printf("Final: ");
	for(i=0; i<MD5_DIGEST_LENGTH; i++)
		printf("%02x", (unsigned char) hash_table[i]);
	printf("\n");
	printf("Time taken by %d threads is %f", NTHREAD, elapsed1+elapsed2);
	hipFree(d_data);
	hipFree(d_hash);
	return 0;
}

